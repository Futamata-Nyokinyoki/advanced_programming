#include "hip/hip_runtime.h"
#include "imageUtil.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <glob.h>

__global__ void kernelGray(Image src, Image templ, Point *position, double *distance, int *is_found)
{
	// Calculate the unique x, y coordinates for this thread
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Check if the coordinates are within the valid range
	if (x < src.width - templ.width && y < src.height - templ.height)
	{
		if (isMatchGray(&src, &templ, x, y))
		{
			position->x = x;
			position->y = y;
			*distance = 0;
			*is_found = 1;
			return;
		}
	}
}

void templateMatchingGray(Image *src, Image *templ, Point *position, double *distance, int *is_found)
{
	Image d_img;
	d_img.channel = src->channel;
	d_img.height = src->height;
	d_img.width = src->width;
	size_t size = src->height * src->width * src->channel * sizeof(unsigned char);
	hipMalloc((void **)&d_img.data, size);
	hipMemcpy(d_img.data, src->data, size, hipMemcpyHostToDevice);

	Image d_templ;
	d_templ.channel = templ->channel;
	d_templ.height = templ->height;
	d_templ.width = templ->width;
	size = templ->width * templ->height * templ->channel * sizeof(unsigned char);
	hipMalloc((void **)&d_templ.data, size);
	hipMemcpy(d_templ.data, templ->data, size, hipMemcpyHostToDevice);

	Point *d_position;
	hipMalloc((void **)&d_position, sizeof(Point));

	double *d_distance;
	hipMalloc((void **)&d_distance, sizeof(double));

	int *d_is_found;
	hipMalloc((void **)&d_is_found, sizeof(int));

	if (src->channel != 3 || templ->channel != 3)
	{
		// デバイス関数では printf は可能だが fprintf は不可能
		printf("src and/or template image is not a color image.\n");
		return;
	}

	int BLOCK_SIZE = 16;
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((src->width - templ->width + dimBlock.x - 1) / dimBlock.x, (src->height - templ->width + dimBlock.y - 1) / dimBlock.y);

	kernelGray<<<dimGrid, dimBlock>>>(d_img, d_templ, d_position, d_distance, d_is_found);

	hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "%s: %s\n", __func__, hipGetErrorString(error));
	}
	// GPUメモリをCPUメモリにコピー
	hipMemcpy(distance, d_distance, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(position, d_position, sizeof(Point), hipMemcpyDeviceToHost);
	hipMemcpy(is_found, d_is_found, sizeof(int), hipMemcpyDeviceToHost);

	// GPUメモリの解放
	hipFree(d_img.data);
	hipFree(d_templ.data);
	hipFree(d_position);
	hipFree(d_distance);
	hipFree(d_is_found);
}

__device__ int isMatchGray(Image *src, Image *templ, int x, int y)
{
	for (int j = 0; j < templ->height; j++)
	{
		for (int i = 0; i < templ->width; i++)
		{
			int pt = (y + j) * src->width + (x + i);
			int pt2 = j * templ->width + i;
			if (src->data[pt] != templ->data[pt2])
			{
				return 0;
			}
		}
	}
	return 1;
}

__global__ void kernelColor(Image src, Image templ, Point *position, double *distance, int *is_found)
{
	// Calculate the unique x, y coordinates for this thread
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Check if the coordinates are within the valid range
	if (x < src.width - templ.width && y < src.height - templ.height)
	{
		if (isMatchColor(&src, &templ, x, y))
		{
			position->x = x;
			position->y = y;
			*distance = 0;
			*is_found = 1;
			return;
		}
	}
}

void templateMatchingColor(Image *src, Image *templ, Point *position, double *distance, int *is_found)
{
	Image d_img;
	d_img.channel = src->channel;
	d_img.height = src->height;
	d_img.width = src->width;
	size_t size = src->height * src->width * src->channel * sizeof(unsigned char);
	hipMalloc((void **)&d_img.data, size);
	hipMemcpy(d_img.data, src->data, size, hipMemcpyHostToDevice);

	Image d_templ;
	d_templ.channel = templ->channel;
	d_templ.height = templ->height;
	d_templ.width = templ->width;
	size = templ->width * templ->height * templ->channel * sizeof(unsigned char);
	hipMalloc((void **)&d_templ.data, size);
	hipMemcpy(d_templ.data, templ->data, size, hipMemcpyHostToDevice);

	Point *d_position;
	hipMalloc((void **)&d_position, sizeof(Point));

	double *d_distance;
	hipMalloc((void **)&d_distance, sizeof(double));

	int *d_is_found;
	hipMalloc((void **)&d_is_found, sizeof(int));

	if (src->channel != 3 || templ->channel != 3)
	{
		// デバイス関数では printf は可能だが fprintf は不可能
		printf("src and/or template image is not a color image.\n");
		return;
	}

	int BLOCK_SIZE = 32;
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((src->width - templ->width + dimBlock.x - 1) / dimBlock.x, (src->height - templ->width + dimBlock.y - 1) / dimBlock.y);

	kernelColor<<<dimGrid, dimBlock>>>(d_img, d_templ, d_position, d_distance, d_is_found);

	hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "%s: %s\n", __func__, hipGetErrorString(error));
	}
	// GPUメモリをCPUメモリにコピー
	hipMemcpy(distance, d_distance, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(position, d_position, sizeof(Point), hipMemcpyDeviceToHost);
	hipMemcpy(is_found, d_is_found, sizeof(int), hipMemcpyDeviceToHost);

	// GPUメモリの解放
	hipFree(d_img.data);
	hipFree(d_templ.data);
	hipFree(d_position);
	hipFree(d_distance);
	hipFree(d_is_found);
}

__device__ int isMatchColor(Image *src, Image *templ, int x, int y)
{
	for (int j = 0; j < templ->height; j++)
	{
		for (int i = 0; i < templ->width; i++)
		{
			int pt = 3 * ((y + j) * src->width + (x + i));
			int pt2 = 3 * (j * templ->width + i);
			if (src->data[pt + 0] != templ->data[pt2 + 0] ||
					src->data[pt + 1] != templ->data[pt2 + 1] ||
					src->data[pt + 2] != templ->data[pt2 + 2])
			{
				return 0;
			}
		}
	}
	return 1;
}

// test/beach3.ppm template /airgun_women_syufu.ppm 0 0.5 cwp
int level1(char *input_file, char *templ_file, int rotation, double threshold, char *options)
{

	char output_name_base[256];
	char output_name_txt[256];
	char output_name_img[256];
	strcpy(output_name_base, "result/");
	strcat(output_name_base, getBaseName(input_file));
	strcpy(output_name_txt, output_name_base);
	strcat(output_name_txt, ".txt");
	strcpy(output_name_img, output_name_base);

	int isWriteImageResult = 0;
	int isPrintResult = 0;
	int isGray = 0;

	if (options != NULL)
	{
		char *p = NULL;
		if ((p = strchr(options, 'c')) != NULL)
			clearResult(output_name_txt);
		if ((p = strchr(options, 'w')) != NULL)
			isWriteImageResult = 1;
		if ((p = strchr(options, 'p')) != NULL)
			isPrintResult = 1;
		if ((p = strchr(options, 'g')) != NULL)
			isGray = 1;
	}

	Image *img = readPXM(input_file);
	Image *templ = readPXM(templ_file);

	Point result;
	double distance = INT_MAX;
	int is_found = 0;
	// 初期化終了

	// テンプレートマッチング開始

	// hipMalloc((void **)&d_img, sizeof(Image));
	// hipMalloc((void **)&d_templ, sizeof(Image));

	if (isGray && img->channel == 3)
	{
		Image *img_gray = createImage(img->width, img->height, 1);
		Image *templ_gray = createImage(templ->width, templ->height, 1);
		cvtColorGray(img, img_gray);
		cvtColorGray(templ, templ_gray);

		// テンプレートマッチング
		templateMatchingGray(img, templ, &result, &distance, &is_found);

		freeImage(img_gray);
		freeImage(templ_gray);
	}
	else
	{
		// テンプレートマッチング
		templateMatchingColor(img, templ, &result, &distance, &is_found);
	}

	// 後処理開始
	if (is_found)
	{
		writeResult(output_name_txt, getBaseName(templ_file), result, templ->width, templ->height, rotation, distance);
		if (isPrintResult)
		{
			// printf("[Found    ] %s %d %d %d %d %d %f\n", getBaseName(templ_file), result.x, result.y, templ->width, templ->height, rotation, distance);
		}
		if (isWriteImageResult)
		{
			drawRectangle(img, result, templ->width, templ->height);

			if (img->channel == 3)
				strcat(output_name_img, ".ppm");
			else if (img->channel == 1)
				strcat(output_name_img, ".pgm");
			writePXM(output_name_img, img);
		}
	}
	else
	{
		if (isPrintResult)
		{
			// printf("[Not found] %s %d %d %d %d %d %f\n", getBaseName(templ_file), result.x, result.y, templ->width, templ->height, rotation, distance);
		}
	}

	freeImage(img);
	freeImage(templ);

	// 後処理終了

	return 0;
}

void process_image(char *image, char *level)
{
	char *bname = getBaseName(image);
	strcat(bname, ".ppm");
	char *name = (char *)malloc(256);
	strcpy(name, "imgproc/");
	strcat(name, bname);

	int rotation = 0;

	glob_t glob_result;
	char *search_path = (char *)malloc(256);
	strcpy(search_path, level);
	strcat(search_path, "/*.ppm");
	glob(search_path, GLOB_TILDE, NULL, &glob_result);

	int x = 0;
	for (unsigned int i = 0; i < glob_result.gl_pathc; i++)
	{
		char *templ = glob_result.gl_pathv[i];
		if (x == 0)
		{
			level1(name, templ, rotation, 0.5, "cwp");
			x = 1;
		}
		else
		{
			level1(name, templ, rotation, 0.5, "wp");
		}
	}
	globfree(&glob_result);
}

int main(int argc, char *argv[])
{
	char *level = argv[1];
	glob_t glob_result;

	char input_path[256];
	strcpy(input_path, level);
	strcat(input_path, "/final/*.ppm");

	glob(input_path, GLOB_TILDE, NULL, &glob_result);
	for (unsigned int i = 0; i < glob_result.gl_pathc; ++i)
	{
		char *image = glob_result.gl_pathv[i];
		process_image(image, level);
	}
	globfree(&glob_result);
	return 0;
}