#include "imageUtil.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
// #include <omp.h>

void templateMatchingGray(Image *src, Image *templ, Point *position, double *distance)
{
	if (src->channel != 1 || templ->channel != 1)
	{
		fprintf(stderr, "src and/or templeta image is not a gray image.\n");
		return;
	}

	for (int y = 0; y < src->height - templ->height; y++)
	{
		for (int x = 0; x < src->height - templ->height; x++)
		{
			if (isMatchGray(src, templ, x, y))
			{
				position->x = x;
				position->y = y;
				*distance = 0;
				return;
			}
		}
	}

	position->x = 0;
	position->y = 0;
	*distance = INT_MAX;
	return;
}

int isMatchGray(Image *src, Image *templ, int x, int y)
{
	for (int j = 0; j < templ->height; j++)
	{
		for (int i = 0; i < templ->width; i++)
		{
			int pt = (y + j) * src->width + (x + i);
			int pt2 = j * templ->width + i;
			if (src->data[pt] != templ->data[pt2])
			{
				return 0;
			}
		}
	}
	return 1;
}

void templateMatchingColor(Image *src, Image *templ, Point *position, double *distance)
{
	if (src->channel != 3 || templ->channel != 3)
	{
		fprintf(stderr, "src and/or template image is not a color image.\n");
		return;
	}

	for (int y = 0; y < src->height - templ->height; y++)
	{
		for (int x = 0; x < src->width - templ->width; x++)
		{
			if (isMatchColor(src, templ, x, y))
			{
				position->x = x;
				position->y = y;
				*distance = 0;
				return;
			}
		}
	}

	position->x = 0;
	position->y = 0;
	*distance = INT_MAX;
	return;
}

int isMatchColor(Image *src, Image *templ, int x, int y)
{
	for (int j = 0; j < templ->height; j++)
	{
		for (int i = 0; i < templ->width; i++)
		{
			int pt = 3 * ((y + j) * src->width + (x + i));
			int pt2 = 3 * (j * templ->width + i);
			if (src->data[pt + 0] != templ->data[pt2 + 0] ||
					src->data[pt + 1] != templ->data[pt2 + 1] ||
					src->data[pt + 2] != templ->data[pt2 + 2])
			{
				return 0;
			}
		}
	}
	return 1;
}

// test/beach3.ppm template /airgun_women_syufu.ppm 0 0.5 cwp
int main(int argc, char **argv)
{
	CalcTime t;
	// 初期化
	t.start();
	if (argc < 5)
	{
		fprintf(stderr, "Usage: templateMatching src_image temlate_image rotation threshold option(c,w,p,g)\n");
		fprintf(stderr, "Option:\nc) clear a txt result. \nw) write result a image with rectangle.\np) print results.\n");
		fprintf(stderr, "ex: templateMatching src_image.ppm temlate_image.ppm 0 1.0  \n");
		fprintf(stderr, "ex: templateMatching src_image.ppm temlate_image.ppm 0 1.0 c\n");
		fprintf(stderr, "ex: templateMatching src_image.ppm temlate_image.ppm 0 1.0 w\n");
		fprintf(stderr, "ex: templateMatching src_image.ppm temlate_image.ppm 0 1.0 p\n");
		fprintf(stderr, "ex: templateMatching src_image.ppm temlate_image.ppm 0 1.0 g\n");
		fprintf(stderr, "ex: templateMatching src_image.ppm temlate_image.ppm 0 1.0 cw\n");
		fprintf(stderr, "ex: templateMatching src_image.ppm temlate_image.ppm 0 1.0 cwp\n");
		fprintf(stderr, "ex: templateMatching src_image.ppm temlate_image.ppm 0 1.0 cwpg\n");
		return -1;
	}

	char *input_file = argv[1];
	char *templ_file = argv[2];
	int rotation = atoi(argv[3]);
	double threshold = atof(argv[4]);

	// printf("rotation -> %d\n", rotation);

	char output_name_base[256];
	char output_name_txt[256];
	char output_name_img[256];
	strcpy(output_name_base, "result/");
	strcat(output_name_base, getBaseName(input_file));
	strcpy(output_name_txt, output_name_base);
	strcat(output_name_txt, ".txt");
	strcpy(output_name_img, output_name_base);

	int isWriteImageResult = 0;
	int isPrintResult = 0;
	int isGray = 0;

	if (argc == 6)
	{
		char *p = NULL;
		if ((p = strchr(argv[5], 'c')) != NULL)
			clearResult(output_name_txt);
		if ((p = strchr(argv[5], 'w')) != NULL)
			isWriteImageResult = 1;
		if ((p = strchr(argv[5], 'p')) != NULL)
			isPrintResult = 1;
		if ((p = strchr(argv[5], 'g')) != NULL)
			isGray = 1;
	}

	Image *img = readPXM(input_file);
	Image *templ = readPXM(templ_file);

	Point result;
	double distance = 0.0;
	// 初期化終了
	t.end();
	printf("画像：%s\n", input_file);
	printf("テンプレートファイル：%s\n", templ_file);
	printf("初期化. %5.2lf[ms]\n", t.getAvgTime(false));

	// テンプレートマッチング開始
	t.start();
	if (isGray && img->channel == 3)
	{
		Image *img_gray = createImage(img->width, img->height, 1);
		Image *templ_gray = createImage(templ->width, templ->height, 1);
		cvtColorGray(img, img_gray);
		cvtColorGray(templ, templ_gray);

		templateMatchingGray(img_gray, templ_gray, &result, &distance);
	}
	else
	{
		templateMatchingColor(img, templ, &result, &distance);
	}
	// テンプレートマッチング終了
	t.end();
	printf("メイン. %5.2lf[ms]\n", t.getAvgTime(false));

	// 後処理開始
	t.start();
	if (distance < threshold)
	{
		writeResult(output_name_txt, getBaseName(templ_file), result, templ->width, templ->height, rotation, distance);
		if (isPrintResult)
		{
			// printf("[Found    ] %s %d %d %d %d %d %f\n", getBaseName(templ_file), result.x, result.y, templ->width, templ->height, rotation, distance);
		}
		if (isWriteImageResult)
		{
			drawRectangle(img, result, templ->width, templ->height);

			if (img->channel == 3)
				strcat(output_name_img, ".ppm");
			else if (img->channel == 1)
				strcat(output_name_img, ".pgm");
			// printf("out: %s", output_name_img);
			writePXM(output_name_img, img);
		}
	}
	else
	{
		if (isPrintResult)
		{
			// printf("[Not found] %s %d %d %d %d %d %f\n", getBaseName(templ_file), result.x, result.y, templ->width, templ->height, rotation, distance);
		}
	}

	freeImage(img);
	freeImage(templ);

	// 後処理終了
	t.end();
	printf("後処理. %5.2lf[ms]\n", t.getAvgTime(false));

	printf("\n");

	return 0;
}
