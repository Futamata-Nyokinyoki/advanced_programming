#include "imageUtil.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <glob.h>
// #include <omp.h>

void templateMatchingGray(Image *src, Image *templ, Point *position, double *distance)
{
	if (src->channel != 1 || templ->channel != 1)
	{
		fprintf(stderr, "src and/or templeta image is not a gray image.\n");
		return;
	}

	for (int y = 0; y < src->height - templ->height; y++)
	{
		for (int x = 0; x < src->height - templ->height; x++)
		{
			if (isMatchGray(src, templ, x, y))
			{
				position->x = x;
				position->y = y;
				*distance = 0;
				return;
			}
		}
	}

	position->x = 0;
	position->y = 0;
	*distance = INT_MAX;
	return;
}

int isMatchGray(Image *src, Image *templ, int x, int y)
{
	for (int j = 0; j < templ->height; j++)
	{
		for (int i = 0; i < templ->width; i++)
		{
			int pt = (y + j) * src->width + (x + i);
			int pt2 = j * templ->width + i;
			if (src->data[pt] != templ->data[pt2])
			{
				return 0;
			}
		}
	}
	return 1;
}

void templateMatchingColor(Image *src, Image *templ, Point *position, double *distance)
{
	if (src->channel != 3 || templ->channel != 3)
	{
		fprintf(stderr, "src and/or template image is not a color image.\n");
		return;
	}

	for (int y = 0; y < src->height - templ->height; y++)
	{
		for (int x = 0; x < src->width - templ->width; x++)
		{
			if (isMatchColor(src, templ, x, y))
			{
				position->x = x;
				position->y = y;
				*distance = 0;
				return;
			}
		}
	}

	position->x = 0;
	position->y = 0;
	*distance = INT_MAX;
	return;
}

int isMatchColor(Image *src, Image *templ, int x, int y)
{
	for (int j = 0; j < templ->height; j++)
	{
		for (int i = 0; i < templ->width; i++)
		{
			int pt = 3 * ((y + j) * src->width + (x + i));
			int pt2 = 3 * (j * templ->width + i);
			if (src->data[pt + 0] != templ->data[pt2 + 0] ||
					src->data[pt + 1] != templ->data[pt2 + 1] ||
					src->data[pt + 2] != templ->data[pt2 + 2])
			{
				return 0;
			}
		}
	}
	return 1;
}

// test/beach3.ppm template /airgun_women_syufu.ppm 0 0.5 cwp
int level1(char *input_file, char *templ_file, int rotation, double threshold, char *options)
{
	CalcTime t;
	// 初期化
	t.start();

	char output_name_base[256];
	char output_name_txt[256];
	char output_name_img[256];
	strcpy(output_name_base, "result/");
	strcat(output_name_base, getBaseName(input_file));
	strcpy(output_name_txt, output_name_base);
	strcat(output_name_txt, ".txt");
	strcpy(output_name_img, output_name_base);

	int isWriteImageResult = 0;
	int isPrintResult = 0;
	int isGray = 0;

	if (options != NULL)
	{
		char *p = NULL;
		if ((p = strchr(options, 'c')) != NULL)
			clearResult(output_name_txt);
		if ((p = strchr(options, 'w')) != NULL)
			isWriteImageResult = 1;
		if ((p = strchr(options, 'p')) != NULL)
			isPrintResult = 1;
		if ((p = strchr(options, 'g')) != NULL)
			isGray = 1;
	}

	Image *img = readPXM(input_file);
	Image *templ = readPXM(templ_file);

	Point result;
	double distance = 0.0;
	// 初期化終了
	t.end();
	printf("画像：%s\n", input_file);
	printf("テンプレートファイル：%s\n", templ_file);
	printf("初期化. %5.2lf[ms]\n", t.getAvgTime(false));

	// テンプレートマッチング開始
	t.start();
	if (isGray && img->channel == 3)
	{
		Image *img_gray = createImage(img->width, img->height, 1);
		Image *templ_gray = createImage(templ->width, templ->height, 1);
		cvtColorGray(img, img_gray);
		cvtColorGray(templ, templ_gray);

		templateMatchingGray(img_gray, templ_gray, &result, &distance);
	}
	else
	{
		templateMatchingColor(img, templ, &result, &distance);
	}
	// テンプレートマッチング終了
	t.end();
	printf("メイン. %5.2lf[ms]\n", t.getAvgTime(false));

	// 後処理開始
	t.start();
	if (distance < threshold)
	{
		writeResult(output_name_txt, getBaseName(templ_file), result, templ->width, templ->height, rotation, distance);
		if (isPrintResult)
		{
			// printf("[Found    ] %s %d %d %d %d %d %f\n", getBaseName(templ_file), result.x, result.y, templ->width, templ->height, rotation, distance);
		}
		if (isWriteImageResult)
		{
			drawRectangle(img, result, templ->width, templ->height);

			if (img->channel == 3)
				strcat(output_name_img, ".ppm");
			else if (img->channel == 1)
				strcat(output_name_img, ".pgm");
			// printf("out: %s", output_name_img);
			writePXM(output_name_img, img);
		}
	}
	else
	{
		if (isPrintResult)
		{
			// printf("[Not found] %s %d %d %d %d %d %f\n", getBaseName(templ_file), result.x, result.y, templ->width, templ->height, rotation, distance);
		}
	}

	freeImage(img);
	freeImage(templ);

	// 後処理終了
	t.end();
	printf("後処理. %5.2lf[ms]\n", t.getAvgTime(false));

	printf("\n");

	return 0;
}

void process_image(char *image, char *level)
{
	char *bname = getBaseName(image);
	strcat(bname, ".ppm");
	char *name = (char *)malloc(256);
	strcpy(name, "imgproc/");
	strcat(name, bname);
	printf("name: %s\n", name);

	int rotation = 0;

	glob_t glob_result;
	char *search_path = (char *)malloc(256);
	strcpy(search_path, level);
	strcat(search_path, "/*.ppm");
	glob(search_path, GLOB_TILDE, NULL, &glob_result);

	int x = 0;
	for (unsigned int i = 0; i < glob_result.gl_pathc; i++)
	{
		char *templ = glob_result.gl_pathv[i];
		if (x == 0)
		{
			level1(name, templ, rotation, 0.5, "cwp");
			x = 1;
		}
		else
		{
			level1(name, templ, rotation, 0.5, "wp");
		}
	}
	globfree(&glob_result);
}

int main(int argc, char *argv[])
{
	char *level = argv[1];
	glob_t glob_result;

	char input_path[256];
	strcpy(input_path, level);
	strcat(input_path, "/final/*.ppm");
	printf("test file path: %s\n", input_path);

	glob(input_path, GLOB_TILDE, NULL, &glob_result);
	for (unsigned int i = 0; i < glob_result.gl_pathc; ++i)
	{
		char *image = glob_result.gl_pathv[i];
		printf("image: %s\n", image);
		process_image(image, level);
	}
	globfree(&glob_result);
	return 0;
}