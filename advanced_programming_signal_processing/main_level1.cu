#include "hip/hip_runtime.h"
#include "imageUtil.cuh"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <unistd.h>
#include <glob.h>

__global__ void kernelGray(Image src, Image templ, Point *position, double *distance)
{

	for (int y = 0; y < src.height - templ.height; y++)
	{
		for (int x = 0; x < src.height - templ.height; x++)
		{
			if (isMatchGray(&src, &templ, x, y))
			{
				position->x = x;
				position->y = y;
				*distance = 0;
				return;
			}
		}
	}

	position->x = 0;
	position->y = 0;
	*distance = INT_MAX;
	return;
}

void templateMatchingGray(Image *src, Image *templ, Point *position, double *distance)
{
	Image d_img;
	d_img.channel = src->channel;
	d_img.height = src->height;
	d_img.width = src->width;
	size_t size = src->height * src->width * src->channel * sizeof(unsigned char);
	hipMalloc((void **)&d_img.data, size);
	hipMemcpy(d_img.data, src->data, size, hipMemcpyHostToDevice);

	Image d_templ;
	d_templ.channel = templ->channel;
	d_templ.height = templ->height;
	d_templ.width = templ->width;
	size = templ->width * templ->height * templ->channel * sizeof(unsigned char);
	hipMalloc((void **)&d_templ.data, size);
	hipMemcpy(d_templ.data, templ->data, size, hipMemcpyHostToDevice);

	Point *d_position;
	hipMalloc((void **)&d_position, sizeof(Point));

	d_position->x = position->x;
	d_position->y = position->y;

	double *d_distance;
	hipMalloc((void **)d_distance, sizeof(double));

	if (src->channel != 1 || templ->channel != 1)
	{
		// デバイス関数では printf は可能だが fprintf は不可能
		printf("src and/or templete image is not a gray image.\n");
		return;
	}
	kernelGray<<<1, 1>>>(d_img, d_templ, d_position, d_distance);

	hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "%s: %s\n", __func__, hipGetErrorString(error));
	}

	// GPUメモリをCPUメモリにコピー
	hipMemcpy(distance, &d_distance, sizeof(double), hipMemcpyDeviceToHost);

	printf("%s distance: %lf\n", __func__, distance);

	// GPUメモリの解放
	hipFree(d_img.data);
	hipFree(d_templ.data);
	hipFree(d_position);
	hipFree(d_distance);
	return;
}

__device__ int isMatchGray(Image *src, Image *templ, int x, int y)
{
	for (int j = 0; j < templ->height; j++)
	{
		for (int i = 0; i < templ->width; i++)
		{
			int pt = (y + j) * src->width + (x + i);
			int pt2 = j * templ->width + i;
			if (src->data[pt] != templ->data[pt2])
			{
				return 0;
			}
		}
	}
	return 1;
}

__global__ void kernelColor(Image src, Image templ, Point *position, double *distance)
{

	for (int y = 0; y < src.height - templ.height; y++)
	{
		for (int x = 0; x < src.height - templ.height; x++)
		{
			if (isMatchColor(&src, &templ, x, y))
			{
				position->x = x;
				position->y = y;
				*distance = 0;
				return;
			}
		}
	}

	position->x = 0;
	position->y = 0;
	*distance = INT_MAX;
	return;
}

void templateMatchingColor(Image *src, Image *templ, Point *position, double *distance)
{
	Image d_img;
	d_img.channel = src->channel;
	d_img.height = src->height;
	d_img.width = src->width;
	size_t size = src->height * src->width * src->channel * sizeof(unsigned char);
	hipMalloc((void **)&d_img.data, size);
	hipMemcpy(d_img.data, src->data, size, hipMemcpyHostToDevice);

	Image d_templ;
	d_templ.channel = templ->channel;
	d_templ.height = templ->height;
	d_templ.width = templ->width;
	size = templ->width * templ->height * templ->channel * sizeof(unsigned char);
	hipMalloc((void **)&d_templ.data, size);
	hipMemcpy(d_templ.data, templ->data, size, hipMemcpyHostToDevice);

	Point *d_position;
	hipMalloc((void **)&d_position, sizeof(Point));

	double *d_distance;
	hipMalloc((void **)&d_distance, sizeof(double));

	if (src->channel != 3 || templ->channel != 3)
	{
		// デバイス関数では printf は可能だが fprintf は不可能
		printf("src and/or template image is not a color image.\n");
		return;
	}
	kernelColor<<<1, 1>>>(d_img, d_templ, d_position, d_distance);

	// hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "%s: %s\n", __func__, hipGetErrorString(error));
	}
	// GPUメモリをCPUメモリにコピー
	hipMemcpy(distance, &d_distance, sizeof(double), hipMemcpyDeviceToHost);

	printf("%s distance: %lf\n", __func__, distance);

	// GPUメモリの解放
	hipFree(d_img.data);
	hipFree(d_templ.data);
	hipFree(d_position);
	hipFree(d_distance);
	return;
}

__device__ int isMatchColor(Image *src, Image *templ, int x, int y)
{
	for (int j = 0; j < templ->height; j++)
	{
		for (int i = 0; i < templ->width; i++)
		{
			int pt = 3 * ((y + j) * src->width + (x + i));
			int pt2 = 3 * (j * templ->width + i);
			if (src->data[pt + 0] != templ->data[pt2 + 0] ||
					src->data[pt + 1] != templ->data[pt2 + 1] ||
					src->data[pt + 2] != templ->data[pt2 + 2])
			{
				return 0;
			}
		}
	}
	return 1;
}

// test/beach3.ppm template /airgun_women_syufu.ppm 0 0.5 cwp
int level1(char *input_file, char *templ_file, int rotation, double threshold, char *options)
{
	CalcTime t;
	// 初期化
	t.start();

	char output_name_base[256];
	char output_name_txt[256];
	char output_name_img[256];
	strcpy(output_name_base, "result/");
	strcat(output_name_base, getBaseName(input_file));
	strcpy(output_name_txt, output_name_base);
	strcat(output_name_txt, ".txt");
	strcpy(output_name_img, output_name_base);

	int isWriteImageResult = 0;
	int isPrintResult = 0;
	int isGray = 0;

	if (options != NULL)
	{
		char *p = NULL;
		if ((p = strchr(options, 'c')) != NULL)
			clearResult(output_name_txt);
		if ((p = strchr(options, 'w')) != NULL)
			isWriteImageResult = 1;
		if ((p = strchr(options, 'p')) != NULL)
			isPrintResult = 1;
		if ((p = strchr(options, 'g')) != NULL)
			isGray = 1;
	}

	Image *img = readPXM(input_file);
	Image *templ = readPXM(templ_file);

	Point result;
	double distance = 0.0;
	// 初期化終了
	t.end();
	printf("画像：%s\n", input_file);
	printf("テンプレートファイル：%s\n", templ_file);
	printf("初期化. %5.2lf[ms]\n", t.getAvgTime(false));

	// テンプレートマッチング開始
	t.start();

	// hipMalloc((void **)&d_img, sizeof(Image));
	// hipMalloc((void **)&d_templ, sizeof(Image));

	if (isGray && img->channel == 3)
	{
		Image *img_gray = createImage(img->width, img->height, 1);
		Image *templ_gray = createImage(templ->width, templ->height, 1);
		cvtColorGray(img, img_gray);
		cvtColorGray(templ, templ_gray);

		// // 画像サイズの取得
		// size_t size_img = img_gray->width * img_gray->height * img_gray->channel;
		// size_t size_templ = templ_gray->width * templ_gray->height * templ_gray->channel;

		// // 画像データのメモリ領域確保
		// hipMalloc((void **)&d_img->data, size_img);
		// hipMalloc((void **)&d_templ->data, size_templ);

		// // 画像をGPUにコピー
		// hipMemcpy(d_img->data, img_gray->data, size_img, hipMemcpyHostToDevice);
		// hipMemcpy(d_templ->data, img_gray->data, size_templ, hipMemcpyHostToDevice);

		// // 構造体をGPUにコピー
		// hipMemcpy(d_img, img_gray, sizeof(Image), hipMemcpyHostToDevice);
		// hipMemcpy(d_templ, templ_gray, sizeof(Image), hipMemcpyHostToDevice);

		// テンプレートマッチング
		templateMatchingGray(img, templ, &result, &distance);

		freeImage(img_gray);
		freeImage(templ_gray);
	}
	else
	{

		// // 画像サイズの取得
		// size_t size_img = img->width * img->height * img->channel * sizeof(unsigned char);
		// size_t size_templ = templ->width * templ->height * templ->channel * sizeof(unsigned char);

		// printf("hoge1\n");
		// // 画像データのメモリ領域確保
		// hipMalloc((void **)&d_img->data, size_img);
		// hipMalloc((void **)&d_templ->data, size_templ);

		// printf("hoge2\n");
		// // 画像をGPUにコピー
		// hipMemcpy(d_img->data, img->data, size_img, hipMemcpyHostToDevice);
		// hipMemcpy(d_templ->data, templ->data, size_templ, hipMemcpyHostToDevice);

		// printf("hoge3\n");
		// // 構造体をGPUにコピー
		// hipMemcpy(d_img, img, sizeof(Image), hipMemcpyHostToDevice);
		// hipMemcpy(d_templ, templ, sizeof(Image), hipMemcpyHostToDevice);

		// printf("hoge4\n");
		// テンプレートマッチング
		templateMatchingColor(img, templ, &result, &distance);
		printf("%s distance: %lf\n", __func__, distance);

		// printf("hoge5\n");
		// hipDeviceSynchronize();

		// hipError_t error = hipGetLastError();
		// if (error != hipSuccess)
		// {
		// 	fprintf(stderr, "Error: %s\n", hipGetErrorString(error));
		// }

		// hipMemcpy(img, d_img, sizeof(Image), hipMemcpyDeviceToHost);
		// hipMemcpy(templ, d_templ, sizeof(Image), hipMemcpyDeviceToHost);

		// hipFree(d_img);
		// hipFree(d_templ);
	}
	// テンプレートマッチング終了
	t.end();
	printf("メイン. %5.2lf[ms]\n", t.getAvgTime(false));

	// 後処理開始
	t.start();
	if (distance < threshold)
	{
		printf("hoge1\n");
		writeResult(output_name_txt, getBaseName(templ_file), result, templ->width, templ->height, rotation, distance);
		printf("hoge2\n");
		if (isPrintResult)
		{
			// printf("[Found    ] %s %d %d %d %d %d %f\n", getBaseName(templ_file), result.x, result.y, templ->width, templ->height, rotation, distance);
		}
		if (isWriteImageResult)
		{
			printf("hoge3\n");
			printf("result x: %d\n", result.x);
			printf("result y: %d\n", result.y);
			drawRectangle(img, result, templ->width, templ->height);
			printf("hoge4\n");

			if (img->channel == 3)
				strcat(output_name_img, ".ppm");
			else if (img->channel == 1)
				strcat(output_name_img, ".pgm");
			// printf("out: %s", output_name_img);
			writePXM(output_name_img, img);
		}
	}
	else
	{
		if (isPrintResult)
		{
			// printf("[Not found] %s %d %d %d %d %d %f\n", getBaseName(templ_file), result.x, result.y, templ->width, templ->height, rotation, distance);
		}
	}

	freeImage(img);
	freeImage(templ);

	// 後処理終了
	t.end();
	printf("後処理. %5.2lf[ms]\n", t.getAvgTime(false));

	printf("\n");

	return 0;
}

void process_image(char *image, char *level)
{
	char *bname = getBaseName(image);
	strcat(bname, ".ppm");
	char *name = (char *)malloc(256);
	strcpy(name, "imgproc/");
	strcat(name, bname);
	printf("name: %s\n", name);

	int rotation = 0;

	glob_t glob_result;
	char *search_path = (char *)malloc(256);
	strcpy(search_path, level);
	strcat(search_path, "/*.ppm");
	glob(search_path, GLOB_TILDE, NULL, &glob_result);

	int x = 0;
	for (unsigned int i = 0; i < glob_result.gl_pathc; i++)
	{
		char *templ = glob_result.gl_pathv[i];
		if (x == 0)
		{
			level1(name, templ, rotation, 0.5, "cwp");
			x = 1;
		}
		else
		{
			level1(name, templ, rotation, 0.5, "wp");
		}
	}
	globfree(&glob_result);
}

int main(int argc, char *argv[])
{
	char pathname[256];
	getcwd(pathname, 256);
	printf("pathname: %s\n", pathname);
	char *level = argv[1];
	glob_t glob_result;

	char input_path[256];
	strcpy(input_path, level);
	strcat(input_path, "/final/*.ppm");

	glob(input_path, GLOB_TILDE, NULL, &glob_result);
	for (unsigned int i = 0; i < glob_result.gl_pathc; ++i)
	{
		char *image = glob_result.gl_pathv[i];
		printf("image: %s\n", image);
		process_image(image, level);
	}
	globfree(&glob_result);
	return 0;
}